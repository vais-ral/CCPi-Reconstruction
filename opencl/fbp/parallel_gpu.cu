#include "hip/hip_runtime.h"
#include "defs_gpu.cuh"

#include "time_stamp.h"

texture<float, 1, hipReadModeElementType> texC;
texture<float, 1, hipReadModeElementType> texA;
texture<float, 1, hipReadModeElementType> texX;
texture<float, 1, hipReadModeElementType> texY;

texture<float, 1, hipReadModeElementType> texCos;
texture<float, 2, hipReadModeElementType> texAR;

__global__ void find_cart_new(float * pc_res_d, float *pc_a_d, float *pc_r_d, int cart_len){
	int start = blockIdx.x*cart_len;
	for(int i=start;i<(start+cart_len);i++){
		//pc_res_d[i]= tex2D(texPC,pc_a_d[i],pc_r_d[i]);
		pc_res_d[i]= tex2D(texAR,pc_a_d[i]+0.5f,pc_r_d[i]+0.5f);
	}
}



static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
{
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}


__global__ void cuda_data_r2c(Complex *dev_inc, float *dev_in, unsigned int sp){
	unsigned int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = i + sp;
	dev_inc[i].x = dev_in[j];
	dev_inc[i].y = 0.0f;
}


__global__ void cuda_data_c2r_3(float *dev_in, Complex *dev_inc, unsigned int shift, unsigned int nc, unsigned int nr, unsigned int nrow){
	unsigned int i, i2, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	i2 = i + shift;
	j =  blockIdx.y * blockDim.y + threadIdx.y;
	dev_in[(nrow+j)*nr+i] = dev_inc[j*nc+i2].x;
}



__global__ void cuda_data_c2r(float *dev_in, Complex *dev_inc, unsigned int sp){
	unsigned int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = i + sp;
	dev_in[j] = dev_inc[i].x;
}

__global__ void cuda_mul_c(const Complex *dev_fc, Complex *dev_inc){
	unsigned int i;
	Complex c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	c = ComplexMul(dev_fc[i],dev_inc[i]);
	dev_inc[i] = c;
}


__global__ void cprod(Complex* a, const Complex* b)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	a[i] = ComplexMul(a[i], b[i]);
}

__global__ void creal(const Complex* a, float* b)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	b[i] = a[i].x;
} 


__global__ void cuda_filter_r2c(Complex *dev_fc, float *dev_fr, unsigned int nx){
	unsigned int i, j, k;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = i%nx;
	k = blockIdx.y * nx + i;
	dev_fc[k].x = dev_fr[j];
	dev_fc[k].y = 0.0f;
}



__global__ void fbp_axial(float* dev_out, float da, unsigned int nx, unsigned int ny, float fnr){
	unsigned int ixx, i;
	float r, s, sum;
		
	ixx = blockIdx.x*blockDim.x + threadIdx.x;
	r = float(ixx)-fnr;
	sum = 0.0f;
			
	for(i = 0; i<ny; i++){
		s = fnr + r*cosf(da*float(i));
		sum += tex1Dfetch(texA,s);
	}

	dev_out[ixx] = sum;
}


__global__ void set_zero(float *veco, unsigned int nxo){	
	unsigned int tig;
	tig = (blockIdx.y*blockDim.y + threadIdx.y)*nxo + blockIdx.x*blockDim.x + threadIdx.x;
	veco[tig] = 0.0f;
}

bool get_gpu_info(allData *aD){
	

/*	
	aD->gi->major = 1;
	aD->gi->minor = 1;
	aD->gi->multiProcessorCount = 16;
	aD->gi->regsPerBlock = 8*1024;
	aD->gi->warpSize = 32;
	aD->gi->sharedMemPerBlock = 16*1024;
//	aD->gi->maxResidentThreads = prop.maxThreadsPerMultiProcessor;//
	aD->gi->maxThreadsPerBlock = 512;

	aD->gi->maxResidentBlocks = 8;
	if(aD->gi->major == 2){
		aD->gi->maxResidentThreads = 1536;//new
		aD->gi->sharedMemBanks = 32;
		aD->gi->maxResidentWarps = 48;
	}else if(aD->gi->minor > 1){
		aD->gi->maxResidentThreads = 1024;//new
		aD->gi->sharedMemBanks = 16;
		aD->gi->maxResidentWarps = 32;
	}else{
		aD->gi->maxResidentThreads = 768;//new
		aD->gi->sharedMemBanks = 16;
		aD->gi->maxResidentWarps = 24;
	}
*/
	/*aD->gi->major = 2;
	aD->gi->minor = 0;
	aD->gi->multiProcessorCount = 30;
	aD->gi->regsPerBlock = 32*1024;
	aD->gi->warpSize = 32;
	aD->gi->sharedMemPerBlock = 48*1024;
//	aD->gi->maxResidentThreads = prop.maxThreadsPerMultiProcessor;//
	aD->gi->maxThreadsPerBlock = 1024;

	aD->gi->maxResidentBlocks = 8;
	if(aD->gi->major == 2){
		aD->gi->maxResidentThreads = 1536;//new
		aD->gi->sharedMemBanks = 32;
		aD->gi->maxResidentWarps = 48;
	}else if(aD->gi->minor > 1){
		aD->gi->maxResidentThreads = 1024;//new
		aD->gi->sharedMemBanks = 16;
		aD->gi->maxResidentWarps = 32;
	}else{
		aD->gi->maxResidentThreads = 768;//new
		aD->gi->sharedMemBanks = 16;
		aD->gi->maxResidentWarps = 24;
	}*/
	
	hipError_t ce;
	int deviceCount;
        int MyDevice;

	printTagStart(aD,"GPUDevices");
	
	ce = hipGetDeviceCount(&deviceCount);
	if(ce != hipSuccess){
		sprintf(aD->message, "can not count GPUs: \"%s\"", hipGetErrorString(ce));
		printError(aD);
		return false;
	}else{
		printTag(aD,"NumberOfDevices",deviceCount);
	}

	if(deviceCount < 1 || deviceCount > 10){
		printError(aD,"device count - wrong number");
		return false;
	}
	
	printTag(aD,"RequestedIndexOfDevice",aD->hms->fbp->GPUDeviceNumber);
        hipGetDevice(&MyDevice);
	printTag(aD,"MyCudaDevice",MyDevice);
	
	hipDeviceProp_t prop;



	hipGetDeviceProperties(&prop,MyDevice);


	aD->gi->major = (unsigned int)prop.major;
	aD->gi->minor = (unsigned int)prop.minor;
	aD->gi->multiProcessorCount = (unsigned int)prop.multiProcessorCount;
	aD->gi->regsPerBlock = (unsigned int)prop.regsPerBlock;
	aD->gi->warpSize = (unsigned int)prop.warpSize;
	aD->gi->sharedMemPerBlock = (unsigned int)(prop.sharedMemPerBlock);
//	aD->gi->maxResidentThreads = prop.maxThreadsPerMultiProcessor;//
	aD->gi->maxThreadsPerBlock = (unsigned int)prop.maxThreadsPerBlock;

	aD->gi->maxResidentBlocks = 8;
	if(aD->gi->major == 2){
		aD->gi->maxResidentThreads = 1536;//new
		aD->gi->sharedMemBanks = 32;
		aD->gi->maxResidentWarps = 48;
	}else if(aD->gi->minor > 1){
		aD->gi->maxResidentThreads = 1024;//new
		aD->gi->sharedMemBanks = 16;
		aD->gi->maxResidentWarps = 32;
	}else{
		aD->gi->maxResidentThreads = 768;//new
		aD->gi->sharedMemBanks = 16;
		aD->gi->maxResidentWarps = 24;
	}

	printTagStart(aD,"GPUDevice");
	printTag(aD,"Name",prop.name);
	printTag(aD,"MultiProcessorCount",(unsigned int)prop.multiProcessorCount);
	printTag(aD,"ClockRate",prop.clockRate/1000,"in MHz");
	printTag(aD,"TotalGlobalMemory",(unsigned int)(prop.totalGlobalMem),"bytes");
	printTag(aD,"TotalGlobalMemory",(float)(float(prop.totalGlobalMem)/1073741824.0),"GB");
	printTag(aD,"RevisionMajor",(unsigned int)prop.major);
	printTag(aD,"RevisionMinor",(unsigned int)prop.minor);

	sprintf(aD->message, "%i x %i x %i", (unsigned int)prop.maxGridSize[0], (unsigned int)prop.maxGridSize[1], (unsigned int)prop.maxGridSize[2]);
	printTag(aD,"MaximumGridSize",aD->message,"maximum size of a grid of thread blocks");

	sprintf(aD->message, "%i x %i x %i", (unsigned int)prop.maxThreadsDim[0], (unsigned int)prop.maxThreadsDim[1], (unsigned int)prop.maxThreadsDim[2]);
	printTag(aD,"MaximumThreadSize",aD->message,"the maximum size of each dimension of a block");

	printTag(aD,"WarpSize",(unsigned int)(prop.warpSize));
	printTag(aD,"MaxWarpsPerMultiProcessor",(unsigned int)(aD->gi->maxResidentWarps),"maximum number of resident warps per multiprocessor");
	
	printTag(aD,"MaxThreadsPerBlock",(unsigned int)(prop.maxThreadsPerBlock));
//	printTag(aD,"MaxThreadsPerMultiProcessor",prop.maxThreadsPerMultiProcessor);

	printTag(aD,"MaxBlocksPerMultiProcessor",(unsigned int)(aD->gi->maxResidentBlocks),"maximum number of resident blocks per multiprocessor");

	printTag(aD,"RegistersPerBlock",(unsigned int)(prop.regsPerBlock),"maximum number of 32-bit registers available to a thread block");
	printTag(aD,"SharedMemoryPerBlock",(unsigned int)(prop.sharedMemPerBlock),"maximum amount of shared memory available to a thread block in bytes");
	printTag(aD,"SharedMemoryBanks",(unsigned int)(aD->gi->sharedMemBanks),"number of shared memory banks");

//	printTag(aD,"L2cache",prop.l2CacheSize,"bytes");

	printTagEnd(aD);//GPUDevice
	printTagEnd(aD);//GPUDevices
	
	return true;
}




bool fbp_axial_cuda(allData *aD){
	gpu_info *gi;
	xData *xd;
	
	unsigned int mb, mt, nx, ny;
	unsigned int uu, ub, maxt;
	float fnr, da;
	size_t mem_size, mem_size_c;
		
	float *dev_in, *dev_out;
	hipfftComplex *dev_filter, *dev_data;

	hipfftHandle plan;
        timestamp("Starting fbp_axial_cuda",4);

	xd = aD->data;
	gi = aD->gi;

	nx = aD->ta_nx;
	ny = aD->ny;

	mb = gi->maxResidentBlocks;
	mt = gi->maxResidentThreads;
	
	maxt = mt/mb;
	
	uu = 1;
	while(maxt>1){
		uu*=2;
		maxt/=2;	
	}
	maxt = uu;
	ub = nx/maxt;

	da = aD->gi->rotAngleStep;

	fnr = 0.5f*float(nx-1);

	dim3 grids_p(ub,1,1);
	dim3 threads_p(maxt,1,1);
          

	mem_size = nx*sizeof(float);
	mem_size_c = nx*sizeof(hipfftComplex);
	
        timestamp("calling cuda to automatically get a device",4);
        hipFree(NULL);
	// hipSetDevice(aD->hms->fbp->GPUDeviceNumber);
         {
		int MyDevice;
		char  devmsg[128];
                hipGetDevice(&MyDevice);
		snprintf(devmsg,128,"my device %i",MyDevice);
		timestamp(devmsg,4);
          }

        timestamp("calling hipMalloc",4);
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_filter,mem_size_c));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_data,mem_size_c));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_in,mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_out,mem_size));

        timestamp("finished hipMalloc",4);
	
	CUDA_SAFE_CALL(hipMemcpy(dev_filter,xd->veccF,mem_size_c,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_data,xd->veccI,mem_size_c,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	HMFFT_SAFE_CALL(hipfftPlan1d(&plan, nx, HIPFFT_C2C, 1));

	HMFFT_SAFE_CALL(hipfftExecC2C(plan, dev_data, dev_data, HIPFFT_FORWARD));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	cprod<<<grids_p,threads_p>>>(dev_data,dev_filter);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	HMFFT_SAFE_CALL(hipfftExecC2C(plan, dev_data, dev_data, HIPFFT_BACKWARD));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	creal<<<grids_p,threads_p>>>(dev_data,dev_in);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	hipfftDestroy(plan);

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	texA.normalized = false;
   	texA.filterMode = hipFilterModeLinear;
   	texA.addressMode[0] = hipAddressModeClamp;

	CUDA_SAFE_CALL(hipBindTexture(NULL, texA, dev_in, channelDesc, mem_size));
	
	fbp_axial<<<grids_p, threads_p>>>(dev_out, da, nx, ny, fnr);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(hipMemcpy(xd->vecto, dev_out, mem_size, hipMemcpyDeviceToHost));

	hipFree(dev_in); dev_in = NULL;
	hipFree(dev_out); dev_out = NULL;
	hipFree(dev_data); dev_data = NULL;
	hipFree(dev_filter); dev_filter = NULL;
        timestamp("Finishing fbp_axial_cuda",4);
				
	return true;
}





__global__ void fbp_std2(float* temp_dev, unsigned int nxo, unsigned int mw, float xc, float *dev_CS, unsigned int vH, unsigned int chnum){
	extern __shared__ float shm[];
	float2 *vcs = (float2 *)&shm[0];
	float sum, p, fitt;
	float x, y;
	unsigned int ixx, iyy, itt, j, tib, tpb;

	tib = threadIdx.y * blockDim.x + threadIdx.x;
	tpb = blockDim.x * blockDim.y;
	ixx = blockIdx.x*blockDim.x + threadIdx.x;
	iyy = blockIdx.y*blockDim.y + threadIdx.y;
	itt = ixx+iyy*nxo;
	fitt = float(itt);
	
	x = tex1Dfetch(texX,fitt);
	y = tex1Dfetch(texY,fitt);
		
	sum = 0.0f;
	
	shm[tib] = dev_CS[tib+chnum*tpb];
	__syncthreads();
	for(j=0; j<vH; j++){
		p = xc + x*vcs[j].x+ y*vcs[j].y + float(j*mw);
		sum += (tex1Dfetch(texA,p));	
	}
	
	temp_dev[itt] += sum;
}


bool fbp_cuda(allData *aD){
	xData *xd = aD->data;

	size_t mem_CS;
	size_t mem_out, mem_in;
	size_t mem_chunk;
	size_t mem_comp;
	size_t mem_shared;

	float *dev_in, *dev_out;
	float *dev_X, *dev_Y;
	float *dev_CS;
		
	unsigned int hTimer;
	unsigned int nx, ny, nxo, nyo, wo, ho;
	unsigned int cl, cw;
	unsigned int vH, sp;
		
	hipfftComplex *dev_fc, *dev_inc;
	hipfftHandle plan;
	double gpuTime;
	
	xFBP_gpu *fg;
        timestamp("starting fbp_cuda",4);
	fg = aD->fg;

	nx = fg->nx;
	ny = fg->ny;
	nxo = fg->nxo;
	nyo = fg->nyo;
	wo = fg->blockWidth;
	ho = fg->blockHeight;
	cl = fg->chunkLeft;
	cw = fg->chunkWidth;
	vH = fg->vH;
	
	mem_shared = wo*ho*sizeof(float);		
	mem_comp = nx*vH*sizeof(hipfftComplex);
	mem_chunk = cw*vH*sizeof(float);
	mem_CS = 2*ny*sizeof(float);
	mem_out = nxo*nyo*sizeof(float);
	mem_in = nx*ny*sizeof(float);

	dim3 grids_in(nx*vH/(wo*ho),1);
	dim3 threads_in(wo*ho,1);

	dim3 grids_ch(cw/wo,vH/ho,1);
	dim3 threads_ch(wo,ho,1);
	
	dim3 grids_bp(nxo/wo,nyo/ho,1);	
	dim3 threads_bp(wo,ho,1);
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	
	CUT_SAFE_CALL( cutCreateTimer(&hTimer) );
	CUT_SAFE_CALL( cutResetTimer(hTimer));
    CUT_SAFE_CALL( cutStartTimer(hTimer));

        timestamp("calling hipMalloc",4);
         {
		int MyDevice;
		char  devmsg[128];
                hipGetDevice(&MyDevice);
		snprintf(devmsg,128,"my device %i",MyDevice);
		timestamp(devmsg,4);
          }
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_fc,mem_comp));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_inc,mem_comp));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_in,mem_in));
        timestamp("finished hipMalloc",4);

	CUDA_SAFE_CALL(hipMemcpy(dev_fc,xd->veccF,mem_comp,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_in,xd->vta,mem_in,hipMemcpyHostToDevice));

	CUFFT_SAFE_CALL(hipfftPlan1d(&plan, nx, HIPFFT_C2C, vH));

	texA.normalized = false;
   	texA.filterMode = hipFilterModeLinear;
   	texA.addressMode[0] = hipAddressModeClamp;

	texX.normalized = false;
   	texX.filterMode = hipFilterModeLinear;
   	texX.addressMode[0] = hipAddressModeClamp;

	texY.normalized = false;
   	texY.filterMode = hipFilterModeLinear;
   	texY.addressMode[0] = hipAddressModeClamp;


        timestamp("calling hipMalloc",4);
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_CS,mem_CS));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_out,mem_out));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_X,mem_out));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_Y,mem_out));
        timestamp("finished hipMalloc",4);

	CUDA_SAFE_CALL(hipMemcpy(dev_X,xd->vecX,mem_out,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_Y,xd->vecY,mem_out,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_CS,xd->vecCS,mem_CS,hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipBindTexture(NULL, texA, dev_in, channelDesc, mem_chunk));	
	CUDA_SAFE_CALL(hipBindTexture(NULL, texX, dev_X, channelDesc, mem_out));
	CUDA_SAFE_CALL(hipBindTexture(NULL, texY, dev_Y, channelDesc, mem_out));



	set_zero<<<grids_bp, threads_bp>>>(dev_out, nxo);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	
	for(unsigned int i=0;i<fg->numChunks; i++){
		sp = i * nx * vH;
		cuda_data_r2c<<<grids_in,threads_in>>>(dev_inc,dev_in,sp);
		CUDA_SAFE_CALL( hipDeviceSynchronize() );
		CUFFT_SAFE_CALL(hipfftExecC2C(plan, dev_inc, dev_inc, HIPFFT_FORWARD));
		CUDA_SAFE_CALL( hipDeviceSynchronize() );
		
		cuda_mul_c<<<grids_in,threads_in>>>(dev_fc,dev_inc);
		CUDA_SAFE_CALL( hipDeviceSynchronize() );
	
		CUFFT_SAFE_CALL(hipfftExecC2C(plan, dev_inc, dev_inc, HIPFFT_BACKWARD));
		CUDA_SAFE_CALL( hipDeviceSynchronize() );
		cuda_data_c2r_3<<<grids_ch,threads_ch>>>(dev_in, dev_inc, cl, nx, cw, 0);
		
		CUDA_SAFE_CALL( hipDeviceSynchronize() );	
		
		fbp_std2<<<grids_bp, threads_bp, mem_shared>>>(dev_out, nxo, cw, fg->xc, dev_CS, vH, i);
		CUDA_SAFE_CALL( hipDeviceSynchronize() );
	}

	hipfftDestroy(plan);
	hipFree(dev_inc); dev_inc = NULL;
	hipFree(dev_fc); dev_fc = NULL;

	CUDA_SAFE_CALL(hipMemcpy(xd->vto,dev_out,mem_out,hipMemcpyDeviceToHost));
	
	hipFree(dev_in); dev_in = NULL;
	hipFree(dev_out); dev_out = NULL;
	hipFree(dev_CS); dev_CS = NULL;
	hipFree(dev_X); dev_X = NULL;
	hipFree(dev_Y); dev_Y = NULL;
	
	CUT_SAFE_CALL( cutStopTimer(hTimer) );
    gpuTime = cutGetTimerValue(hTimer);
	printTag(aD,"BackprojectionTime",(float)gpuTime,"in ms");
        timestamp("finished fbp_cuda",4);
    
	return true;
}


__global__ void xb_proj_new(unsigned int indexs, unsigned int starty, float *vif, float *vecxy, float *vec_sc, float xc, unsigned int vH){
	unsigned int ij, ijk, tib, i;
	float s;
	float2 cs, xy;
	
	tib = (blockDim.x*threadIdx.y + threadIdx.x)/2;
	cs = *(float2 *)(&vec_sc[2*(tib+starty)]);
		
	ij = 2*blockIdx.x + indexs;
	for(i = 0; i<2; i++){	
		xy = *(float2 *)(&vecxy[2*ij]);	
		ijk = ij*vH+tib;
		s = __fmaf_rn(xy.x,cs.x,xc);
		vif[ijk] =  __fmaf_rn(xy.y,cs.y,s);
		ij++;
	}

}


__global__ void xb_proj2_new(unsigned int starty, float *vec_x, float *vecxy, float *vec_cs){
	unsigned int ij, ijk, tib, i, tpb;
	float s;
	float2 cs, xy;

	tpb = blockDim.x * blockDim.y;
	tib = (blockDim.x*threadIdx.y + threadIdx.x)/2;
	cs = *(float2 *)(&vec_cs[2*(tib+starty)]);
		
	ij = 2*blockIdx.x;
	for(i = 0; i<2; i++){
		xy = *(float2 *)(&vecxy[2*ij]);	
		ijk = tib*tpb+ij;
		s = __fmul_rn(xy.x,cs.x);
		vec_x[ijk] = __fmaf_rn(xy.y,cs.y,s);
		ij++;
	}
}


__global__ void fbp_std5(float *vb, float *veco, float *vt, unsigned int nx, unsigned int vH, unsigned int nxo, unsigned int nxb){
	float rr, sum;
	
	unsigned int ixq, ivc, ic;
	unsigned int tib, tig, big, tpb;
		
	tib = blockDim.x*threadIdx.y + threadIdx.x;
	big = blockIdx.y*nxb + blockIdx.x;
	tig = (blockIdx.y*blockDim.y + threadIdx.y)*nxo + blockIdx.x*blockDim.x + threadIdx.x;
	tpb = blockDim.x * blockDim.y;
	
	ixq = tib;
	ivc = big*vH;
	sum = 0.0f;	
	
	for(ic = 0; ic<vH; ic++){		
		rr = __fadd_rn(vt[ixq],vb[ivc+ic]);
		sum += tex1Dfetch(texA,rr+float(ic*nx));	
		ixq += tpb;
	}
	
	veco[tig] += sum;
}


bool fbp_cuda_20(allData *aD){
	xData *xd = aD->data;

	unsigned int nx, ny, nxo, nyo;
	unsigned int wo, ho;
	unsigned int nxb, nyb;
	
	unsigned int mt, mb;
	unsigned int uu, ub;
	unsigned int maxt;
	unsigned int vH;
	unsigned int nf, sp;
	unsigned int starty, indexs;
	unsigned int nxyb, nlen, nxyb2;

	unsigned int hTimer;

	size_t mem_fc, mem_chunk;
	size_t mem_in, mem_out;
	size_t mem_CS, mem_block_xy;
	size_t mem_thread_x, mem_thread_xy, mem_block;
	
	float xc;

	float *dev_thread_x, *dev_thread_xy, *dev_block;
	float *dev_in, *dev_out;
	float *dev_block_xy;
	float *dev_CS;
	
	gpu_info *gi;
	
	double gpuTime;
			
	hipfftComplex *dev_fc, *dev_inc;

	hipfftHandle plan;
        timestamp("starting fbp_cuda_20",4);

	gi = aD->gi;
	
	xc = aD->new_xc;
	vH = gi->vertH;

	wo = gi->wo;
	ho = gi->ho;
	
	nx = aD->ta_nx;
	ny = aD->ta_ny;
	nxo = gi->mxo;
	nyo = gi->myo;	
	
	nxb = nxo/wo;
	nyb = nyo/ho;

	mem_block = nxb*nyb*vH*sizeof(float); 
	mem_thread_x = wo*ho*vH*sizeof(float);
	mem_thread_xy = 2*wo*ho*sizeof(float);
	mem_block_xy = 2 * nxb * nyb * sizeof(float);

	mem_in = nx * ny *sizeof(float);
	mem_out = nxo * nyo *sizeof(float);
	mem_CS = 2*ny*sizeof(float);
	mem_fc = nx * vH * sizeof(hipfftComplex);
	mem_chunk = nx * vH * sizeof(float);
	
	mb = gi->maxResidentBlocks;
	mt = gi->maxResidentThreads;
	
	maxt = mt/mb;
	
	uu = 1;
	while(maxt>1){
		uu*=2;
		maxt/=2;	
	}
	maxt = uu;
	ub = nx/maxt;

	nf = ny/vH;

	nxyb = nxb*nyb;
	
	nlen = (nxyb/4);
	if(nxyb%4 > 0) nlen++;
	
	nxyb2 = nxyb-2*nlen;

	dim3 threads_xb(1,1,1);
	dim3 threads_1(maxt,1,1);
	dim3 threads_bp(wo,ho,1);

	dim3 grid_xb2(vH,1,1);	
	dim3 grid_xb_new(nlen,1,1);	
	dim3 grids_2(ub*vH,1,1);
	dim3 grid_bp(nxb,nyb,1);	
	
	HMCUT_SAFE_CALL( cutCreateTimer(&hTimer) );
	HMCUT_SAFE_CALL( cutResetTimer(hTimer));
        HMCUT_SAFE_CALL( cutStartTimer(hTimer));
	
        timestamp("calling hipMalloc ",4);
         {
		int MyDevice;
		char  devmsg[128];
                hipGetDevice(&MyDevice);
		snprintf(devmsg,128,"my device %i",MyDevice);
		timestamp(devmsg,4);
          }
	HM_SAFE_CALL(hipMalloc((void**)&dev_block,mem_block));
	HM_SAFE_CALL(hipMalloc((void**)&dev_thread_x,mem_thread_x));
	HM_SAFE_CALL(hipMalloc((void**)&dev_thread_xy,mem_thread_xy));
	HM_SAFE_CALL(hipMalloc((void**)&dev_fc,mem_fc));
	HM_SAFE_CALL(hipMalloc((void**)&dev_inc,mem_fc));
	HM_SAFE_CALL(hipMalloc((void**)&dev_in,mem_in));
	HM_SAFE_CALL(hipMalloc((void**)&dev_out,mem_out));
	HM_SAFE_CALL(hipMalloc((void**)&dev_block_xy,mem_block_xy));
	HM_SAFE_CALL(hipMalloc((void**)&dev_CS,mem_CS));
        timestamp("finished  hipMalloc ",4);

	HM_SAFE_CALL(hipMemcpy(dev_fc,xd->veccF,mem_fc,hipMemcpyHostToDevice));
	HM_SAFE_CALL(hipMemcpy(dev_in,xd->vta,mem_in,hipMemcpyHostToDevice));
	HM_SAFE_CALL(hipMemcpy(dev_block_xy,xd->vecbXY,mem_block_xy,hipMemcpyHostToDevice));
	HM_SAFE_CALL(hipMemcpy(dev_thread_xy,xd->vecXY_block,mem_thread_xy,hipMemcpyHostToDevice));
	HM_SAFE_CALL(hipMemcpy(dev_CS,xd->vecCS,mem_CS,hipMemcpyHostToDevice));

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	texA.normalized = false;
   	texA.filterMode = hipFilterModeLinear;
   	texA.addressMode[0] = hipAddressModeClamp;

	CUDA_SAFE_CALL(hipBindTexture(NULL, texA, dev_in, channelDesc, mem_chunk));	

        /*rca: AARGH -- plan is done for each slice -- this defeats the purpose of planning the FFT */
        timestamp("before plan creation",4);
	HMFFT_SAFE_CALL(hipfftPlan1d(&plan, nx, HIPFFT_C2C, vH));
        timestamp("after plan creation",4);

	set_zero<<<grid_bp, threads_bp>>>(dev_out, nxo);
	HM_SAFE_CALL( hipDeviceSynchronize() );
        timestamp("before backprojection loop ",4);
	
	for(unsigned int i = 0;i<nf;i++){
		sp = i*nx*vH;
		cuda_data_r2c<<<grids_2,threads_1>>>(dev_inc,dev_in,sp);
		HM_SAFE_CALL( hipDeviceSynchronize() );
		HMFFT_SAFE_CALL(hipfftExecC2C(plan, dev_inc, dev_inc, HIPFFT_FORWARD));
		HM_SAFE_CALL( hipDeviceSynchronize() );
		cuda_mul_c<<<grids_2,threads_1>>>(dev_fc,dev_inc);
		
		HM_SAFE_CALL( hipDeviceSynchronize() );
		HMFFT_SAFE_CALL(hipfftExecC2C(plan, dev_inc, dev_inc, HIPFFT_BACKWARD));
		HM_SAFE_CALL( hipDeviceSynchronize() );
		sp = 0;
		cuda_data_c2r<<<grids_2,threads_1>>>(dev_in,dev_inc,sp);
		HM_SAFE_CALL( hipDeviceSynchronize() );

		starty = i*vH;
		indexs = 0;
		xb_proj_new<<<grid_xb_new, threads_bp>>>(indexs, starty, dev_block, dev_block_xy, dev_CS, xc, vH);
		HM_SAFE_CALL( hipDeviceSynchronize() );
		indexs = nxyb2;
		xb_proj_new<<<grid_xb_new, threads_bp>>>(indexs, starty, dev_block, dev_block_xy, dev_CS, xc, vH);
		HM_SAFE_CALL( hipDeviceSynchronize() );

		xb_proj2_new<<<grid_xb2, threads_bp>>>(starty, dev_thread_x, dev_thread_xy, dev_CS);
		HM_SAFE_CALL( hipDeviceSynchronize() );
		fbp_std5<<<grid_bp, threads_bp>>>(dev_block, dev_out, dev_thread_x, nx, vH, nxo, nxb);
		HM_SAFE_CALL( hipDeviceSynchronize() );
	}
        timestamp("after backprojection loop ",4);
	
	hipfftDestroy(plan);
        timestamp("after plan destruction",4);
		
	HM_SAFE_CALL(hipMemcpy(xd->vto, dev_out, mem_out, hipMemcpyDeviceToHost));

	hipFree(dev_fc); dev_fc = NULL;
	hipFree(dev_inc); dev_inc = NULL;
	hipFree(dev_in); dev_in = NULL;
	hipFree(dev_block); dev_block = NULL;
	hipFree(dev_thread_x); dev_thread_x = NULL;
	hipFree(dev_thread_xy); dev_thread_xy = NULL;
	hipFree(dev_out); dev_out = NULL;
	hipFree(dev_CS); dev_CS = NULL;
	hipFree(dev_block_xy); dev_block_xy = NULL;

	HMCUT_SAFE_CALL( cutStopTimer(hTimer) );

        gpuTime = cutGetTimerValue(hTimer);

	printTag(aD,"TimeBackprojection",float(gpuTime),"time (in ms)");
   	
        timestamp("finished fbp_cuda_20",4);
	return true;
}


__global__ void fbp_cpu2(float *dev_in, float *dev_pol, float *dev_Cos, unsigned int starty, unsigned int nx, unsigned int na, unsigned int vH, unsigned int pola, float ps, float xc_new){
	float x, xx, sum;
	
	unsigned int ic, irr;
	unsigned int tib, tig, tpb;
	int iaa, aa, ia, dr, dm;
		
	tib = blockDim.x*threadIdx.y + threadIdx.x;
	iaa = blockIdx.x*blockDim.x + threadIdx.x;
	irr = blockIdx.y*blockDim.y + threadIdx.y;
	tig = irr*pola + iaa;
	tpb = blockDim.x * blockDim.y;

	x = ps * float(irr);
		
	sum = 0.0f;	
	float t;

	ia = starty + iaa;
	dm = ia/na;
	dr = ia - dm * na;
	if(dm%2 == 0){
		t= 1.0f;
	}else{
		t= -1.0f;
	}
	
	for(ic = 0; ic<vH; ic++){
		if(dr == na){
			dr = 0;
			t*= -1.0f;
		}
		
		xx = xc_new + t* x * tex1Dfetch(texCos, dr);
		sum += tex1Dfetch(texA,xx + float(ic*nx));	
		dr++;
		
	}
	
	dev_pol[tig] += sum;
}



bool fbp_cuda_cpu2(allData *aD){
	xData *xd = aD->data;

	unsigned int nx, ny, polr, pola, nba, nbr;
	unsigned int wo, ho;
		
	unsigned int mt, mb;
	unsigned int uu, ub;
	unsigned int maxt;
	unsigned int vH;
	unsigned int nf, sp, na;
	unsigned int starty, indexs;
	
	unsigned int hTimer;

	size_t mem_fc, mem_chunk;
	size_t mem_in;
	size_t mem_Cos, mem_pol;
		
	float xc, ps;
	
	float *dev_in, *dev_pol;
	float *dev_Cos;
	
	gpu_info *gi;
	
	double gpuTime;
        timestamp("calling fbp_cuda_cpu2",4);

	gi = aD->gi;

	pola = gi->pol_a;
	polr = gi->pol_r;

	ps = gi->outputPixelSize;
			
	hipfftComplex *dev_fc, *dev_inc;

	hipfftHandle plan;

	
	
	xc = aD->new_xc;
	vH = gi->vertH;

	wo = gi->wo;
	ho = gi->ho;
	
	nx = aD->ta_nx;
	ny = aD->ta_ny;

	na = aD->ta_ny_13;

	nba = pola/wo;
	nbr = polr/ho;

	printf("pola: %i, polr: %i\n",pola, polr);
		

	mem_in = nx * ny *sizeof(float);
	mem_pol = pola *polr *sizeof(float);
	mem_Cos = na*sizeof(float);
	mem_fc = nx * vH * sizeof(hipfftComplex);
	mem_chunk = nx * vH * sizeof(float);
	
	mb = gi->maxResidentBlocks;
	mt = gi->maxResidentThreads;
	
	maxt = mt/mb;
	
	uu = 1;
	while(maxt>1){
		uu*=2;
		maxt/=2;	
	}
	maxt = uu;
	ub = nx/maxt;

	nf = ny/vH;

	dim3 threads_1(maxt,1,1);
	dim3 threads_bp(wo,ho,1);
	
	dim3 grids_2(ub*vH,1,1);
	dim3 grid_bp(nba,nbr,1);	
	
	HMCUT_SAFE_CALL( cutCreateTimer(&hTimer) );
	HMCUT_SAFE_CALL( cutResetTimer(hTimer));
        HMCUT_SAFE_CALL( cutStartTimer(hTimer));
	
         {
		int MyDevice;
		char  devmsg[128];
                hipGetDevice(&MyDevice);
		snprintf(devmsg,128,"my device %i",MyDevice);
		timestamp(devmsg,4);
          }
	HM_SAFE_CALL(hipMalloc((void**)&dev_fc,mem_fc));
	HM_SAFE_CALL(hipMalloc((void**)&dev_inc,mem_fc));
	HM_SAFE_CALL(hipMalloc((void**)&dev_in,mem_in));
	HM_SAFE_CALL(hipMalloc((void**)&dev_pol,mem_pol));
	HM_SAFE_CALL(hipMalloc((void**)&dev_Cos,mem_Cos));

	HM_SAFE_CALL(hipMemcpy(dev_fc,xd->veccF,mem_fc,hipMemcpyHostToDevice));
	HM_SAFE_CALL(hipMemcpy(dev_in,xd->vta,mem_in,hipMemcpyHostToDevice));
	HM_SAFE_CALL(hipMemcpy(dev_Cos,xd->vecCos,mem_Cos,hipMemcpyHostToDevice));

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	texA.normalized = false;
   	texA.filterMode = hipFilterModeLinear;
   	texA.addressMode[0] = hipAddressModeClamp;

	texCos.normalized = false;
   	texCos.filterMode = hipFilterModeLinear;
   	texCos.addressMode[0] = hipAddressModeClamp;

	CUDA_SAFE_CALL(hipBindTexture(NULL, texA, dev_in, channelDesc, mem_chunk));	
	CUDA_SAFE_CALL(hipBindTexture(NULL, texCos, dev_Cos, channelDesc, mem_Cos));	

	HMFFT_SAFE_CALL(hipfftPlan1d(&plan, nx, HIPFFT_C2C, vH));

	set_zero<<<grid_bp, threads_bp>>>(dev_pol, nba);
	HM_SAFE_CALL( hipDeviceSynchronize() );
	
	for(unsigned int i = 0;i<nf;i++){
	//for(unsigned int i = 0;i<1;i++){
		sp = i*nx*vH;
		cuda_data_r2c<<<grids_2,threads_1>>>(dev_inc,dev_in,sp);
		HM_SAFE_CALL( hipDeviceSynchronize() );
		HMFFT_SAFE_CALL(hipfftExecC2C(plan, dev_inc, dev_inc, HIPFFT_FORWARD));
		HM_SAFE_CALL( hipDeviceSynchronize() );
		cuda_mul_c<<<grids_2,threads_1>>>(dev_fc,dev_inc);
		
		HM_SAFE_CALL( hipDeviceSynchronize() );
		HMFFT_SAFE_CALL(hipfftExecC2C(plan, dev_inc, dev_inc, HIPFFT_BACKWARD));
		HM_SAFE_CALL( hipDeviceSynchronize() );
		sp = 0;
		cuda_data_c2r<<<grids_2,threads_1>>>(dev_in,dev_inc,sp);
		HM_SAFE_CALL( hipDeviceSynchronize() );

		starty = i*vH;
		
		fbp_cpu2<<<grid_bp, threads_bp>>>(dev_in, dev_pol, dev_Cos, starty, nx, na, vH, pola, ps, xc);
		HM_SAFE_CALL( hipDeviceSynchronize() );
	}
	
	hipfftDestroy(plan);
		
	HM_SAFE_CALL(hipMemcpy(xd->vecPol, dev_pol, mem_pol, hipMemcpyDeviceToHost));

	hipFree(dev_fc); dev_fc = NULL;
	hipFree(dev_inc); dev_inc = NULL;
	hipFree(dev_in); dev_in = NULL;
	
	hipFree(dev_pol); dev_pol = NULL;
	hipFree(dev_Cos); dev_Cos = NULL;
	
	HMCUT_SAFE_CALL( cutStopTimer(hTimer) );
    gpuTime = cutGetTimerValue(hTimer);
	printTag(aD,"TimeBackprojection",float(gpuTime),"time (in ms)");
   	
	return true;
}

