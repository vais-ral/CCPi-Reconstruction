#include "hip/hip_runtime.h"
#include "defs_gpu.cuh"


__global__ void find_cart(float * pc_res_d, float *pc_a_d, float *pc_r_d, int cart_len){
	int start = blockIdx.x*cart_len;
	for(int i=start;i<(start+cart_len);i++){
		//pc_res_d[i]= tex2D(texPC,pc_a_d[i],pc_r_d[i]);
		pc_res_d[i]= tex2D(texPC,pc_a_d[i]+0.5f,pc_r_d[i]+0.5f);
	}
}

__global__ void find_fdk2(float* vecv_d, float *m_x_d, float *m_y_d, int *i_a_d, int *i_r_d, int im, int na_old, int nrn, int aimin, int w_a, int w_r, size_t pitch_pc){
	int rr1, aa1;
		
	rr1 = (i_r_d[blockIdx.x]*blockDim.y+threadIdx.y)*w_r;
	aa1 = (i_a_d[blockIdx.x]*blockDim.x+threadIdx.x)*w_a;


	int rr, aa, a2, m1, m2;
	int p4 = pitch_pc/4;
	
	for(int j=0;j<w_a;j++){	
		aa = aa1+j;
		a2 = (aa+im+aimin)%(na_old);
		for(int i=0;i<w_r;i++){
			rr = rr1+i;
			m1 = rr*p4+aa;
			m2 = rr*na_old+a2;
			//vecv_d[m1]+= tex2D(texI,m_x_d[m2],m_y_d[m2]);
			vecv_d[m1]+= tex2D(texI, m_x_d[m2]+0.5f, m_y_d[m2]+0.5f);
			//vecv_d[m1] = m2;
			//vecv_d[m1]= m1;//aa1;
			//vecv_d[m1]= rr1;//1.2+blockDim.y;//i_r_d[blockIdx.x];//tex2D(texI,m_x_d[m2],m_y_d[m2]);
		}
	}
	
}


__global__ void find_fdk3(float* vecv_d, float *m_x_d, float *m_y_d, int *i_a_d, int *i_r_d, int im, int na_old, int nrn, int aimin, int w_a, int w_r, size_t pitch_pc, float vx, float vy){
	int rr1, aa1;
		
	rr1 = (i_r_d[blockIdx.x]*blockDim.y+threadIdx.y)*w_r;
	aa1 = (i_a_d[blockIdx.x]*blockDim.x+threadIdx.x)*w_a;


	int rr, aa, a2, m1, m2;
	int p4 = pitch_pc/4;
	
	for(int j=0;j<w_a;j++){	
		aa = aa1+j;
		a2 = (aa+im+aimin)%(na_old);
		for(int i=0;i<w_r;i++){
			rr = rr1+i;
			m1 = rr*p4+aa;
			m2 = rr*na_old+a2;
			//vecv_d[m1]+= tex2D(texI,m_x_d[m2],m_y_d[m2]);
			vecv_d[m1]+= tex2D(texI, m_x_d[m2]+0.5f-vx, m_y_d[m2]+0.5f-vy);
			//vecv_d[m1] = m2;
			//vecv_d[m1]= m1;//aa1;
			//vecv_d[m1]= rr1;//1.2+blockDim.y;//i_r_d[blockIdx.x];//tex2D(texI,m_x_d[m2],m_y_d[m2]);
		}
	}
	
}



bool cudaReconstructFDK(MatPar *matp, PolarToCart *PC, allData *aD, OtherParam *other){
	int nx, ny;
	nx = aD->nx;
	ny = matp->y_maxi - matp->y_mini+1;

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	hipSetDevice(0);

	int ustep3 = 32;
	dim3 grid_cr(nx/ustep3,ny);
	dim3 grid_cr2(1,ny);

	int volMatrix;
	volMatrix = matp->nr*matp->na;
	printf("volMatrix: %i\n",volMatrix);

	dim3 gfdk2(PC->ntb,1,1);
	dim3 tfdk2(b_a,b_r);
	
	float *veci_d;
	float *m_x_d, *m_y_d;

	int mem_allm = volMatrix*sizeof(float);

	HM_SAFE_CALL(hipMalloc((void**)&m_x_d,mem_allm));
	HM_SAFE_CALL(hipMalloc((void**)&m_y_d,mem_allm));
	HM_SAFE_CALL(hipMemcpy(m_x_d,matp->m_x,mem_allm,hipMemcpyHostToDevice));
	HM_SAFE_CALL(hipMemcpy(m_y_d,matp->m_y,mem_allm,hipMemcpyHostToDevice));

	size_t pitch_pc;
	float *vecv_d;
	HM_SAFE_CALL(hipMallocPitch((void**)&vecv_d, &pitch_pc, PC->nan * sizeof(float), PC->nrn+1));
	HM_SAFE_CALL(hipMemset2D(vecv_d, pitch_pc, 0, PC->nan*sizeof(float), PC->nrn+1));

	texPC.normalized = false;
	texPC.filterMode = hipFilterModeLinear;
	texPC.addressMode[0] = hipAddressModeClamp;
   	texPC.addressMode[1] = hipAddressModeClamp;
	hipChannelFormatDesc channelDesc_pc = hipCreateChannelDesc<float>();
	hipBindTexture2D(NULL, &texPC, vecv_d, &channelDesc_pc, PC->nan, PC->nrn+1, pitch_pc);

	printf("PC_nan: %i\n",PC->nan);
	printf("PC_nrn: %i\n",PC->nrn);
	printf("pitch_pc: %i\n",pitch_pc);

	int mem_size_ntb = PC->ntb*sizeof(Ipp32s);

	int *i_a_d, *i_r_d;
	HM_SAFE_CALL(hipMalloc((void**)&i_a_d,mem_size_ntb));
	HM_SAFE_CALL(hipMalloc((void**)&i_r_d,mem_size_ntb));
	HM_SAFE_CALL(hipMemcpy(i_a_d,PC->i_a,mem_size_ntb,hipMemcpyHostToDevice));
	HM_SAFE_CALL(hipMemcpy(i_r_d,PC->i_r,mem_size_ntb,hipMemcpyHostToDevice));

	size_t pitch;
		
	HM_SAFE_CALL(hipMallocPitch((void**)&veci_d, &pitch, nx * sizeof(float), ny));

	texI.normalized = false;
	texI.filterMode = hipFilterModeLinear;
   	texI.addressMode[0] = hipAddressModeClamp;
   	texI.addressMode[1] = hipAddressModeClamp;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipBindTexture2D(NULL, &texI, veci_d, &channelDesc, nx, ny, pitch);
	
	for(int i=0;i<other->images_to_process;i++){
		HM_SAFE_CALL(hipMemcpy2D(veci_d, pitch, matp->ivol+i*nx*ny,nx*sizeof(float),nx*sizeof(float),ny,hipMemcpyHostToDevice));		
		find_fdk3<<<gfdk2,tfdk2>>>(vecv_d, m_x_d, m_y_d, i_a_d, i_r_d, i, matp->na, PC->nrn, PC->aimin, PC->w_a, PC->w_r, pitch_pc, aD->data->shift_x[i], aD->data->shift_y[i]);
		//find_fdk2<<<gfdk2,tfdk2>>>(vecv_d, m_x_d, m_y_d, i_a_d, i_r_d, i, matp->na, PC->nrn, PC->aimin, PC->w_a, PC->w_r, pitch_pc);
		HM_SAFE_CALL( hipDeviceSynchronize() );
	}

	int mem_cart = PC->pc_len*sizeof(float);

	hipUnbindTexture(&texI);
	HM_SAFE_CALL(hipFree(veci_d)); veci_d = NULL;
	HM_SAFE_CALL(hipFree(m_x_d)); m_x_d = NULL;
	HM_SAFE_CALL(hipFree(m_y_d)); m_y_d = NULL;
	HM_SAFE_CALL(hipFree(i_a_d)); i_a_d = NULL;
	HM_SAFE_CALL(hipFree(i_r_d)); i_r_d = NULL;


	int cart_len = 256;
	dim3 tcart(1,1,1);
	dim3 gcart(PC->pc_len/cart_len,1,1);
	
	float *pc_r_d, *pc_a_d, *pc_res_d;
	
	HM_SAFE_CALL(hipMalloc((void**)&pc_r_d,mem_cart));
	HM_SAFE_CALL(hipMalloc((void**)&pc_a_d,mem_cart));
	HM_SAFE_CALL(hipMalloc((void**)&pc_res_d,mem_cart));
	HM_SAFE_CALL(hipMemcpy(pc_r_d,PC->pc_r,mem_cart,hipMemcpyHostToDevice));
	HM_SAFE_CALL(hipMemcpy(pc_a_d,PC->pc_a,mem_cart,hipMemcpyHostToDevice));
	HM_SAFE_CALL(hipMemcpy(pc_res_d,PC->pc_res,mem_cart,hipMemcpyHostToDevice));
		

	find_cart<<<gcart,tcart>>>(pc_res_d,pc_a_d,pc_r_d,cart_len);
	HM_SAFE_CALL( hipDeviceSynchronize() );
	HM_SAFE_CALL(hipMemcpy(PC->pc_res,pc_res_d,mem_cart,hipMemcpyDeviceToHost));
	HM_SAFE_CALL( hipDeviceSynchronize() );
	hipUnbindTexture(&texPC);

	ippsZero_32f(matp->rec,PC->OutputHeight * PC->OutputWidth);

	int sp;
	sp = 0;
	
	for(int j=0;j<PC->pc_nc;j++){
		ippsCopy_32f(PC->pc_res+sp,matp->rec+PC->pc_pos[j],PC->pc_size[j]);
		sp+=(PC->pc_size[j]);
	}
	
	HM_SAFE_CALL(hipFree(vecv_d)); vecv_d = NULL;
	HM_SAFE_CALL(hipFree(pc_r_d)); pc_r_d = NULL;
	HM_SAFE_CALL(hipFree(pc_a_d)); pc_a_d = NULL;
	HM_SAFE_CALL(hipFree(pc_res_d)); pc_res_d = NULL;

	return true;
}






